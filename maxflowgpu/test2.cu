#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <fstream>
#include <iostream>
#include <sstream>
#include <string>

#define N 5 // Number of nodes (example size)

using namespace std;

// CUDA error checking
#define cudaCheckError() { \
    hipError_t e = hipGetLastError(); \
    if (e != hipSuccess) { \
        printf("CUDA error: %s\n", hipGetErrorString(e)); \
        exit(1); \
    } \
}

void readInput(const char* filename, int total_nodes, int* residual) {
    ifstream file;
    file.open(filename);

    if (!file) {
        cout << "Error reading file!";
        exit(1);
    }

    string line;
    int source, destination;
    int numberOfEdges = 0;
    float capacity;

    while (getline(file, line)) {
        if (line.empty()) continue;

        stringstream linestream(line);
        if (!(linestream >> source >> destination >> capacity)) {
            cerr << "Error parsing line: " << line << endl;
            continue;
        }

        source--;
        destination--;
        int scaledCapacity = static_cast<int>(capacity * 1000);
        if (!residual) {
            cerr << "Memory allocation failed for residual matrix.";
            exit(EXIT_FAILURE);
        }

        numberOfEdges++;
        residual[source * total_nodes + destination] = scaledCapacity;
    }

    cout << "Number of edges in graph is: " << numberOfEdges << endl;
    file.close();
}

__global__ void topDownBFS(int *adjMatrix, bool *frontier, bool *newFrontier, int *visited, int n, int *parent, int *flow) {
    int u = blockIdx.x * blockDim.x + threadIdx.x;
    if (u < n && frontier[u]) {
        for (int v = 0; v < n; ++v) {
            if (adjMatrix[u * n + v] > 0 && !visited[v]) {
                newFrontier[v] = true;
                visited[v] = true;
                parent[v] = u;
                flow[v] = min(flow[u], adjMatrix[u * n + v]); // Calculate flow along the path
            }
        }
    }
}

__global__ void bottomUpBFS(int *adjMatrix, bool *frontier, bool *newFrontier, int *visited, int n, int *parent, int *flow) {
    int v = blockIdx.x * blockDim.x + threadIdx.x;
    if (v < n && !visited[v]) {
        for (int u = 0; u < n; ++u) {
            if (adjMatrix[u * n + v] > 0 && frontier[u]) {
                newFrontier[v] = true;
                visited[v] = true;
                parent[v] = u;
                flow[v] = min(flow[u], adjMatrix[u * n + v]); // Calculate flow along the path
                break;
            }
        }
    }
}

void bfs(int *adjMatrix, int n, int source, int sink, int &maxFlow) {
    bool *frontier, *newFrontier;
    int *visited, *parent, *flow;

    hipMallocManaged(&frontier, n * sizeof(bool));
    hipMallocManaged(&newFrontier, n * sizeof(bool));
    hipMallocManaged(&visited, n * sizeof(int));
    hipMallocManaged(&parent, n * sizeof(int));
    hipMallocManaged(&flow, n * sizeof(int));

    for (int i = 0; i < n; ++i) {
        frontier[i] = false;
        newFrontier[i] = false;
        visited[i] = 0;
        parent[i] = -1;
        flow[i] = 0;
    }
    frontier[source] = true;
    visited[source] = true;
    flow[source] = INT_MAX; // Set initial flow to maximum value

    bool isTopDown = true;
    int frontierSize = 1;

    while (frontierSize > 0 && !visited[sink]) {
        int blockSize = 256;
        int numBlocks = (n + blockSize - 1) / blockSize;

        if (isTopDown) {
            topDownBFS<<<numBlocks, blockSize>>>(adjMatrix, frontier, newFrontier, visited, n, parent, flow);
        } else {
            bottomUpBFS<<<numBlocks, blockSize>>>(adjMatrix, frontier, newFrontier, visited, n, parent, flow);
        }

        hipDeviceSynchronize();
        cudaCheckError();

        // Count new frontier size and decide if we should switch approach
        frontierSize = 0;
        for (int i = 0; i < n; ++i) {
            frontier[i] = newFrontier[i];
            newFrontier[i] = false;
            if (frontier[i]) {
                frontierSize++;
            }
        }

        if (frontierSize > n / 10) { // Example threshold for switching
            isTopDown = !isTopDown;
        }
    }

    // Calculate flow along augmenting path and update residual capacities
    int pathFlow = flow[sink];
    if (pathFlow == 0) return;

    maxFlow += pathFlow;

    int v = sink;
    while (parent[v] != -1) {
        int u = parent[v];
        adjMatrix[u * n + v] -= pathFlow;
        adjMatrix[v * n + u] += pathFlow; // Update backward edge
        v = u;
    }

    hipFree(frontier);
    hipFree(newFrontier);
    hipFree(visited);
    hipFree(parent);
    hipFree(flow);
}

int main(int argc, char* argv[]) {
    if (argc != 2) {
        cout << "Usage: " << argv[0] << " <input_file>" << endl;
        return 1;
    }

    const char* filename = argv[1];

    int *adjMatrix;
    hipMallocManaged(&adjMatrix, N * N * sizeof(int));
    memset(adjMatrix, 0, N * N * sizeof(int)); // Initialize the adjacency matrix with zeros

    readInput(filename, N, adjMatrix);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    int maxFlow = 0;
    while (true) {
        bool *frontier, *newFrontier;
        int *visited, *parent, *flow;
        bool foundPath = false;

        hipMallocManaged(&frontier, N * sizeof(bool));
        hipMallocManaged(&newFrontier, N * sizeof(bool));
        hipMallocManaged(&visited, N * sizeof(int));
        hipMallocManaged(&parent, N * sizeof(int));
        hipMallocManaged(&flow, N * sizeof(int));

        for (int i = 0; i < N; ++i) {
            frontier[i] = false;
            newFrontier[i] = false;
            visited[i] = 0;
            parent[i] = -1;
            flow[i] = 0;
        }
        frontier[0] = true; // Assume source is node 0
        visited[0] = true;
        flow[0] = INT_MAX; // Set initial flow to maximum value

        bool isTopDown = true;
        int frontierSize = 1;

        while (frontierSize > 0 && !visited[N - 1]) { // Assume sink is node N-1
            int blockSize = 256;
            int numBlocks = (N + blockSize - 1) / blockSize;

            if (isTopDown) {
                topDownBFS<<<numBlocks, blockSize>>>(adjMatrix, frontier, newFrontier, visited, N, parent, flow);
                cout << "TD" << endl;
            } else {
                cout << "BU" << endl;
                bottomUpBFS<<<numBlocks, blockSize>>>(adjMatrix, frontier, newFrontier, visited, N, parent, flow);
            }

            hipDeviceSynchronize();
            cudaCheckError();

            // Count new frontier size and decide if we should switch approach
            frontierSize = 0;
            for (int i = 0; i < N; ++i) {
                frontier[i] = newFrontier[i];
                newFrontier[i] = false;
                if (frontier[i]) {
                    frontierSize++;
                }
            }

            if (frontierSize > N / 10) { // Example threshold for switching
                isTopDown = !isTopDown;
            }
        }

        if (!visited[N - 1]) break; // No augmenting path found

        int pathFlow = flow[N - 1];
        maxFlow += pathFlow;

        int v = N - 1;
        while (parent[v] != -1) {
            int u = parent[v];
            adjMatrix[u * N + v] -= pathFlow;
            adjMatrix[v * N + u] += pathFlow; // Update backward edge
            v = u;
        }

        hipFree(frontier);
        hipFree(newFrontier);
        hipFree(visited);
        hipFree(parent);
        hipFree(flow);
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    cout << "Maximum Flow: " << maxFlow << endl;
    cout << "Time elapsed: " << milliseconds << " ms" << endl;

    hipFree(adjMatrix);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    return 0;
}
