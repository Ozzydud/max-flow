#include <iostream>
#include <fstream>
#include <sstream>
#include <string>
#include <vector>
#include <cmath>

#include <bits/stdc++.h>

// CUDA libraries
#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>

using namespace std;

#define INF 1e9


void readInput(const char* filename, int total_nodes, int* residual) {

        ifstream file;
        file.open(filename);

        if (!file) {
        cout <<  "Error reading file!";
        exit(1);
    }

    string line;
    int source, destination;
    int numberOfEdges = 0;

    float capacity;
    cout << "before loop" << endl;
    while (getline(file, line)) {
        if (line.empty()) continue;

        stringstream linestream(line);
        //linestream >> source >> destination >> capacity;
        if (!(linestream >> source >> destination >> capacity)) {
    	cerr << "Error parsing line: " << line << endl;
   	 continue;
	}

	//cout << "reading lines" << endl;
        //cout << "Read: Source=" << source << ", Destination=" << destination << ", Capacity=" << capacity << endl;

        source--;
        destination--;
        //cout << "before scaling" << endl;
        int scaledCapacity = static_cast<int>(capacity * 1000);
        if (!residual) {
    	cerr << "Memory allocation failed for residual matrix.";
    	exit(EXIT_FAILURE);
	}

	numberOfEdges++;
	//cout << "after scaling" << endl;
        residual[source * total_nodes + destination] = scaledCapacity;
        //cout << "adding to residual" << endl;

        //cout << "Residual capacity[" << source << "][" << destination << "]: " << residual[source * total_nodes + destination] << endl;
        //counter++;
        //cout << counter << endl;
       
    }
    
    cout << "Number of edges in graph is: " << numberOfEdges << endl;
    file.close();
}

__global__ void cudaBFS(int* r_capacity, int* parent, int* flow, bool* frontier, bool* visited, int vertices, int source, int* locks) {
    int Idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (!frontier[source] && Idx < vertices && frontier[Idx]) {
        frontier[Idx] = false;
        visited[Idx] = true;

        for (int i = vertices - 1; i >= 0; i--) { 
            if (!frontier[i] && !visited[i] && r_capacity[i * vertices + Idx] > 0) {
                if (atomicCAS(locks + i, 0, 1) == 1 || frontier[i]) {
                    continue;
                }
                frontier[i] = true;
                locks[i] = 0;

                parent[i] = Idx;
                flow[i] = min(flow[Idx], r_capacity[i * vertices + Idx]);
            }
        }
    }
}



__global__ void cudaAugment_path(int* parent, bool* do_change_capacity, int total_nodes, int* r_capacity, int path_flow){
    int Idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(Idx < total_nodes && do_change_capacity[Idx]){
        r_capacity[parent[Idx] * total_nodes + Idx] -= path_flow;
        r_capacity[Idx * total_nodes + parent[Idx]] += path_flow; 
    }    
}


bool sink_reachable(bool* frontier, int total_nodes, int source){
    for (int i = 0; i < total_nodes; i++) {
                if(frontier[i]){
                        return i == source;
                }
        }
        return true;
}




int edmondskarp(const char* filename, int total_nodes) {
    hipError_t cudaStatus = hipSetDevice(4);
    hipEvent_t startEvent3, stopEvent3, startEvent3_1, stopEvent3_1;
    hipEventCreate(&startEvent3);
    hipEventCreate(&stopEvent3);
    hipEventCreate(&startEvent3_1);
    hipEventCreate(&stopEvent3_1);
    float partinitmili = 0.0f;
    float initmili = 0.0f;
    float totalInitTime = 0.0f;
    hipEventRecord(startEvent3);
    if (cudaStatus != hipSuccess) {
        std::cerr << "hipSetDevice failed! Do you have a CUDA-capable GPU installed?";
        return 1;
    }
    // Assuming 3534 or 1107 nodes or 11397 or 39082 or 130228
    int* residual;


    float avgBFSTime = 0;
    int bfsCounter = 0;
    hipEvent_t startEvent, stopEvent;
    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);



    
    hipEvent_t start, stop; // Declare start and stop events
    float milliseconds = 0; // Variable to store elapsed time in milliseconds

    // Initialize CUDA events
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    float avgAUGTime = 0;
    int augCounter = 0;
    hipEvent_t startEvent2, stopEvent2;
    hipEventCreate(&startEvent2);
    hipEventCreate(&stopEvent2);


    // Allocating memory for a square matrix representing the graph
    //residual = (int*)malloc(sizeof(int) * total_nodes * total_nodes);
    cout << "residual" << endl;
    //memset(residual, 0, sizeof(int) * total_nodes * total_nodes);
    cout << "residual1" << endl;

    try {
	residual = new int[total_nodes * total_nodes]();
    } catch (const std::bad_alloc& e) {
	    std::cerr << "Failed to allocate memory for the residual matrix: " << e.what() << std::endl;
	    return 1;
    }





    readInput(filename, total_nodes, residual);
    cout << "data read" << endl;

    int source = 0;
    int sink = total_nodes - 1; // Assuming sink is the last node
    int path_flow;

    int* parent = new int[total_nodes];
    int* flow = new int[total_nodes];
    bool* frontier = new bool[total_nodes];
    bool* visited = new bool [total_nodes];
    bool* do_change_capacity = new bool[total_nodes];
    

    // Set initial flow from source to 0
    flow[source] = 0;
    int* locks = new int[total_nodes];
    int* d_r_capacity, * d_parent, * d_flow, *d_locks;;
    bool* d_frontier, * d_visited, *d_do_change_capacity;

    size_t locks_size = total_nodes * sizeof(int);
    
    cout << "hi1" << endl;
    // Allocate memory on device
    hipMalloc((void**)&d_r_capacity, total_nodes * total_nodes * sizeof(int));
    cout << "hi1" << endl;
    hipMalloc((void**)&d_parent, total_nodes * sizeof(int));
    cout << "hi2" << endl;
    hipMalloc((void**)&d_flow, total_nodes * sizeof(int));
    cout << "hi3" << endl;
    hipMalloc((void**)&d_frontier, total_nodes * sizeof(bool));
    cout << "hi4" << endl;
    hipMalloc((void**)&d_visited, total_nodes * sizeof(bool));
    cout << "hi5" << endl;
    hipMalloc((void**)&d_do_change_capacity, total_nodes * sizeof(bool));
    cout << "hi6" << endl;
    hipMalloc((void**)&d_locks, locks_size);


    // Copy data from host to device
    hipMemcpy(d_r_capacity, residual, total_nodes * total_nodes * sizeof(int), hipMemcpyHostToDevice);


    bool found_augmenting_path;
    int max_flow = 0;
    int block_size = 1024;
    int grid_size = ceil(total_nodes * 1.0 / block_size); //(total_nodes + block_size - 1) / block_size;

    
    cout << "hi1" << endl;
    int counter = 0;
    hipEventRecord(stopEvent3);
    hipEventSynchronize(stopEvent3);
    hipEventElapsedTime(&initmili, startEvent3, stopEvent3);
    totalInitTime += initmili;
    do{
	hipEventRecord(startEvent3_1);
        for (int i = 0; i < total_nodes; ++i) {
        parent[i] = -1; // Initialize parent array
        flow[i] = INF;  // Initialize flow array with INF
        locks[i] = 0;
        if(i == sink){
            frontier[i] = true;
        }else{
            frontier[i] = false;
        }

        visited[i] = false;
        do_change_capacity[i] = false;
        }
   
        hipMemcpy(d_parent, parent, total_nodes * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_flow, flow, total_nodes * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_frontier, frontier, total_nodes * sizeof(bool), hipMemcpyHostToDevice);
        hipMemcpy(d_visited, visited, total_nodes * sizeof(bool), hipMemcpyHostToDevice);
        hipMemcpy(d_locks, locks, locks_size, hipMemcpyHostToDevice);
	    //cout << "hi2" << endl;
	hipEventRecord(stopEvent3_1);
	hipEventSynchronize(stopEvent3_1);
	hipEventElapsedTime(&partinitmili, startEvent3_1, stopEvent3_1);
	totalInitTime += partinitmili;
        while(!sink_reachable(frontier, total_nodes, source)){
	hipEventRecord(startEvent, 0);
        cout << frontier[5] << frontier[6] << frontier[7] << endl;
        // Run BFS kernel
        cudaBFS<<<grid_size, block_size>>>(d_r_capacity, d_parent, d_flow, d_frontier, d_visited, total_nodes, sink, d_locks);
        bfsCounter++;
        // Stop recording the event
        hipEventRecord(stopEvent, 0);
        hipEventSynchronize(stopEvent);

        // Calculate elapsed time
        float bfsmili = 0.0f;
        hipEventElapsedTime(&bfsmili, startEvent, stopEvent);
        avgBFSTime += bfsmili;
        

        hipMemcpy(frontier, d_frontier, total_nodes * sizeof(bool), hipMemcpyDeviceToHost);
        }
        found_augmenting_path = frontier[5];

        if(!found_augmenting_path){
            break;
        }

        hipMemcpy(flow, d_flow, total_nodes * sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(parent, d_parent, total_nodes * sizeof(int), hipMemcpyDeviceToHost);

        path_flow = flow[sink];
        max_flow += path_flow;

        for(int i = sink; i != source; i = parent[i]){
                        do_change_capacity[i] = true;
                }

        hipMemcpy(d_do_change_capacity, do_change_capacity, total_nodes * sizeof(bool), hipMemcpyHostToDevice);

	//cout << "hi4" << endl;
        // Launch BFS kernel
        hipEventRecord(startEvent2, 0);
        cudaAugment_path<<< grid_size, block_size >>>(d_parent, d_do_change_capacity, total_nodes, d_r_capacity, path_flow);

             augCounter++;
        // Stop recording the event
        hipEventRecord(stopEvent2, 0);
        hipEventSynchronize(stopEvent2);

        // Calculate elapsed time
        float augmili = 0.0f;
        hipEventElapsedTime(&augmili, startEvent2, stopEvent2);
        avgAUGTime += augmili;
	//cout << path_flow << endl;
	counter++;
	//cout << "Counter is: " << counter << endl;

    } while(found_augmenting_path); //found_augmenting_path);
    cout << "Counter is: " << counter << endl;
    // Record stop time
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    cout << "Time for BFS and augmenting path: " << milliseconds << " ms\n";
    cout << "Average BFS time is: " << avgBFSTime / bfsCounter << "ms\n";
    cout << "Total time BFS is: " << avgBFSTime << "ms\n";
    cout << "Total AUG time is " << avgAUGTime << "ms\n";
    cout << "Average AUG time is: " << avgAUGTime / augCounter << "ms\n";
    cout << "Total init time is: " << totalInitTime << "ms\n";

    cout << "Maximum Flow: " << max_flow << endl;
    

    // Clean up allocated memory
    delete[] residual;
    delete[] parent;
    delete[] flow;
    delete[] locks;
    delete[] frontier;
    delete[] visited;
    delete[] do_change_capacity;
    hipFree(d_r_capacity);
    hipFree(d_parent);
    hipFree(d_flow);
    hipFree(d_frontier);
    hipFree(d_visited);
    hipFree(d_locks);
    hipFree(d_do_change_capacity);
    // Clean up events
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipEventDestroy(stopEvent);
    hipEventDestroy(startEvent);

    hipEventDestroy(stopEvent2);
    hipEventDestroy(startEvent2);

    hipEventDestroy(stopEvent3);
    hipEventDestroy(startEvent3);

    hipEventDestroy(startEvent3_1);
    hipEventDestroy(stopEvent3_1);

    return 0;
}

int main(){
    cout << "1000x400500" << endl; 
    edmondskarp("data/1000x400500.mtx", 1000);
    cout << "1000x400500 end" << endl; 

    cout << "5000x1250000" << endl; 
    edmondskarp("data/5000x1250000.mtx", 5000);
    cout << "5000x1250000 end" << endl; 

    cout << "10000x2500000" << endl; 
    edmondskarp("data/10000x2500000.mtx", 10000);
    cout << "10000x2500000 end" << endl; 

    
    

    // Assuming 3534 or 1107 nodes or 11397 or 39082 or 130228

}

