#include "hip/hip_runtime.h"
#include <iostream>
#include <limits.h>
#include <queue>
#include <vector>
#include <fstream>

// CUDA libraries
#include <hip/hip_runtime.h>
#include <>

#define V 1106
#define BLOCK_SIZE 256

// CUDA kernel for BFS traversal
__global__ void cuda_bfs(int* rGraph, bool* visited, int* parent, int t, bool* found) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (visited[idx] || *found)
        return;

    visited[idx] = true;
    parent[idx] = -1;

    if (idx == t) {
        *found = true;
        return;
    }

    for (int v = 0; v < V; v++) {
        if (!visited[v] && rGraph[idx][v] > 0) {
            parent[v] = idx;
        }
    }
}

// CUDA kernel for calculating path flow
__global__ void cuda_calculate_path_flow(int* rGraph, int* parent, int* path_flow, int s, int t) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx == t) {
        int v = idx;
        int u = parent[v];
        *path_flow = rGraph[u][v];
        while (u != s) {
            v = u;
            u = parent[v];
            *path_flow = min(*path_flow, rGraph[u][v]);
        }
    }
}

// CUDA kernel for updating residual capacities
__global__ void cuda_update_residual_capacities(int* rGraph, int* parent, int* path_flow, int s, int t) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx == t) {
        int v = idx;
        int u = parent[v];
        while (u != s) {
            v = u;
            u = parent[v];
            rGraph[u][v] -= *path_flow;
            rGraph[v][u] += *path_flow;
        }
    }
}

// Returns the maximum flow from s to t in the given graph
int fordFulkerson(int* graph, int s, int t) {
    int* rGraph;
    hipMalloc(&rGraph, V * V * sizeof(int));
    hipMemcpy(rGraph, graph, V * V * sizeof(int), hipMemcpyHostToDevice);

    bool* visited;
    hipMalloc(&visited, V * sizeof(bool));
    bool* d_found;
    hipMalloc(&d_found, sizeof(bool));
    int* parent;
    hipMalloc(&parent, V * sizeof(int));

    int max_flow = 0;
    bool found_path;

    do {
        found_path = false;
        hipMemset(visited, 0, V * sizeof(bool));
        hipMemset(d_found, false, sizeof(bool));

        cuda_bfs<<<(V + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(rGraph, visited, parent, t, d_found);
        hipDeviceSynchronize();

        bool h_found;
        hipMemcpy(&h_found, d_found, sizeof(bool), hipMemcpyDeviceToHost);

        if (h_found) {
            int* path_flow;
            hipMalloc(&path_flow, sizeof(int));

            cuda_calculate_path_flow<<<1, 1>>>(rGraph, parent, path_flow, s, t);
            hipDeviceSynchronize();

            int h_path_flow;
            hipMemcpy(&h_path_flow, path_flow, sizeof(int), hipMemcpyDeviceToHost);

            cuda_update_residual_capacities<<<1, 1>>>(rGraph, parent, path_flow, s, t);
            hipDeviceSynchronize();

            max_flow += h_path_flow;
            hipFree(path_flow);
        }
    } while (found_path);

    hipFree(rGraph);
    hipFree(visited);
    hipFree(d_found);
    hipFree(parent);

    return max_flow;
}

int main() {
    std::ifstream infile("data/cage3.mtx");
    int* graph = new int[V][V];


    for (int i = 0; i < V && infile; ++i) {
        for (int j = 0; j < V && infile; ++j) {
            infile >> graph[i][j];
        }
    }

    std::cout << "The maximum possible flow is " << graph << std::endl;

    // Check if reading was successful
    if (!infile) {
        std::cerr << "Error reading from file!" << std::endl;
        return 1;
    }

    int max_flow = fordFulkerson(graph, 0, 1000);

    std::cout << "The maximum possible flow is " << max_flow << std::endl;

    delete[] graph;

    return 0;
}
