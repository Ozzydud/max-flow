#include <iostream>
#include <fstream>
#include <sstream>
#include <string>
#include <vector>
#include <cmath>

#include <bits/stdc++.h>

// CUDA libraries
#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>

using namespace std;

#define INF 1e9


void readInput(const char* filename, int total_nodes, int* residual) {

        ifstream file;
        file.open(filename);

        if (!file) {
        cout <<  "Error reading file!";
        exit(1);
    }

    string line;
    int source, destination;
    float capacity;

    while (getline(file, line)) {
        if (line.empty()) continue;

        stringstream linestream(line);
        linestream >> source >> destination >> capacity;

        //cout << "Read: Source=" << source << ", Destination=" << destination << ", Capacity=" << capacity << endl;

        source--;
        destination--;

        int scaledCapacity = static_cast<int>(capacity * 1000);
        residual[source * total_nodes + destination] = scaledCapacity;

        //cout << "Residual capacity[" << source << "][" << destination << "]: " << residual[source * total_nodes + destination] << endl;
    }
    

    file.close();
}

__global__ void cudaBFS(int *r_capacity, int *parent, int *flow, bool *frontier, bool* visited, int vertices, int sink, int* locks){
    int Idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (!frontier[sink] && Idx < vertices && frontier[Idx]) {
        frontier[Idx] = false;
        visited[Idx] = true;

        for (int i = Idx; i < vertices; i++) {
            if (!frontier[i] && !visited[i] && r_capacity[Idx * vertices + i] > 0) {
                if(atomicCAS(locks+i, 0 , 1) == 1 || frontier[i]){
                                continue;
                }
                frontier[i] = true;
                locks[i] = 0;


                parent[i] = Idx;
                flow[i] = min(flow[Idx], r_capacity[Idx * vertices + i]);
            }
        }

        for (int i = 0; i < Idx; i++) {
            if (!frontier[i] && !visited[i] && r_capacity[Idx * vertices + i] > 0) {
                if(atomicCAS(locks+i, 0 , 1) == 1 || frontier[i]){
                                continue;
                }
                frontier[i] = true;
                locks[i] = 0;
                parent[i] = Idx;
                flow[i] = min(flow[Idx], r_capacity[Idx * vertices + i]);
            }
        }
    }
}


__global__ void cudaAugment_path(int* parent, bool* do_change_capacity, int total_nodes, int* r_capacity, int path_flow){
    int Idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(Idx < total_nodes && do_change_capacity[Idx]){
        r_capacity[parent[Idx] * total_nodes + Idx] -= path_flow;
        r_capacity[Idx * total_nodes + parent[Idx]] += path_flow; 
    }    
}


bool sink_reachable(bool* frontier, int total_nodes, int sink){
    for (int i = total_nodes-1; i > -1; --i) {
                if(frontier[i]){
                        return i == sink;
                }
        }
        return true;
}



int main() {
    hipError_t cudaStatus = hipSetDevice(4);
    if (cudaStatus != hipSuccess) {
        std::cerr << "hipSetDevice failed! Do you have a CUDA-capable GPU installed?";
        return 1;
    }
    int total_nodes = 3534; // Assuming 3534 or 1107 nodes or 11397
    int* residual;
    
    hipEvent_t start, stop; // Declare start and stop events
    float milliseconds = 0; // Variable to store elapsed time in milliseconds

    // Initialize CUDA events
    hipEventCreate(&start);
    hipEventCreate(&stop);


    // Allocating memory for a square matrix representing the graph
    residual = (int*)malloc(sizeof(int) * total_nodes * total_nodes);

    memset(residual, 0, sizeof(int) * total_nodes * total_nodes);


    readInput("data/cage9.mtx", total_nodes, residual);
  

    int source = 0;
    int sink = total_nodes - 1; // Assuming sink is the last node
    int path_flow;

    int* parent = new int[total_nodes];
    int* flow = new int[total_nodes];
    bool* frontier = new bool[total_nodes];
    bool* visited = new bool [total_nodes];
    bool* do_change_capacity = new bool[total_nodes];
    

    // Set initial flow from source to 0
    flow[source] = 0;
    int* locks = new int[total_nodes];
    int* d_r_capacity, * d_parent, * d_flow, *d_locks;;
    bool* d_frontier, * d_visited, *d_do_change_capacity;

    size_t locks_size = total_nodes * sizeof(int);
    
    // Allocate memory on device
    hipMalloc((void**)&d_r_capacity, total_nodes * total_nodes * sizeof(int));
    hipMalloc((void**)&d_parent, total_nodes * sizeof(int));
    hipMalloc((void**)&d_flow, total_nodes * sizeof(int));
    hipMalloc((void**)&d_frontier, total_nodes * sizeof(bool));
    hipMalloc((void**)&d_visited, total_nodes * sizeof(bool));
    hipMalloc((void**)&d_do_change_capacity, total_nodes * sizeof(bool));
    hipMalloc((void **)&d_locks, locks_size);


    // Copy data from host to device
    hipMemcpy(d_r_capacity, residual, total_nodes * total_nodes * sizeof(int), hipMemcpyHostToDevice);


    bool found_augmenting_path;
    int max_flow = 0;
    int block_size = 1024;
    int grid_size = ceil(total_nodes * 1.0 / block_size); //(total_nodes + block_size - 1) / block_size;

    hipEventRecord(start);
    cout << "hi1" << endl;
    int counter = 0;

    do{
        for (int i = 0; i < total_nodes; ++i) {
        parent[i] = -1; // Initialize parent array
        flow[i] = INF;  // Initialize flow array with INF
        locks[i] = 0;
        if(i == source){
            frontier[i] = true;
        }else{
            frontier[i] = false;
        }

        visited[i] = false;
        do_change_capacity[i] = false;
        }
   
        hipMemcpy(d_parent, parent, total_nodes * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_flow, flow, total_nodes * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_frontier, frontier, total_nodes * sizeof(bool), hipMemcpyHostToDevice);
        hipMemcpy(d_visited, visited, total_nodes * sizeof(bool), hipMemcpyHostToDevice);
        hipMemcpy(d_locks, locks, locks_size, hipMemcpyHostToDevice);
	//cout << "hi2" << endl;
        while(!sink_reachable(frontier, total_nodes, sink)){
        cudaBFS<<<grid_size, block_size>>>(d_r_capacity,  d_parent, d_flow, d_frontier, d_visited, total_nodes, sink, d_locks);
        //cout << "hi3" << endl;
        

        hipMemcpy(frontier, d_frontier, total_nodes * sizeof(bool), hipMemcpyDeviceToHost);
        }

        found_augmenting_path = frontier[sink];

        if(!found_augmenting_path){
            break;
        }

        hipMemcpy(flow, d_flow, total_nodes * sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(parent, d_parent, total_nodes * sizeof(int), hipMemcpyDeviceToHost);

        path_flow = flow[sink];
        max_flow += path_flow;

        for(int i = sink; i != source; i = parent[i]){
                        do_change_capacity[i] = true;
            i--;
                }

        hipMemcpy(d_do_change_capacity, do_change_capacity, total_nodes * sizeof(bool), hipMemcpyHostToDevice);

	//cout << "hi4" << endl;
        // Launch BFS kernel
        cudaAugment_path<<< grid_size, block_size >>>(d_parent, d_do_change_capacity, total_nodes, d_r_capacity, path_flow);
	cout << path_flow << endl;
	counter++;
	cout << "Counter is: " << counter << endl;

    } while(counter != 16); //found_augmenting_path);
    cout << "hi6" << endl;
    // Record stop time
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    cout << "Time for BFS and augmenting path: " << milliseconds << " ms\n";

    cout << "Maximum Flow: " << max_flow << endl;
    

    // Clean up allocated memory
    delete[] residual;
    delete[] parent;
    delete[] flow;
    delete[] locks;
    hipFree(d_r_capacity);
    hipFree(d_parent);
    hipFree(d_flow);
    hipFree(frontier);
    hipFree(visited);
    hipFree(d_locks);
    // Clean up events
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}


