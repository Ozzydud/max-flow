#include <iostream>
#include <fstream>
#include <vector>

// CUDA libraries
#include <hip/hip_runtime.h>


#define V 5
#define BLOCK_SIZE 5

// CUDA kernel for BFS traversal
__global__ void cuda_bfs(int* rGraph, bool* visited, int* parent, int t, bool* found) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (visited[idx] || *found)
        return;

    visited[idx] = true;
    parent[idx] = -1;

    if (idx == t) {
        *found = true;
        return;
    }

    for (int v = 0; v < V; v++) {
        if (!visited[v] && rGraph[idx * V + v] > 0) {
            parent[v] = idx;
        }
    }
}

// CUDA kernel for calculating path flow
__global__ void cuda_calculate_path_flow(int* rGraph, int* parent, int* path_flow, int s, int t) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx == t) {
        int v = idx;
        int u = parent[v];
        *path_flow = rGraph[u * V + v];
        while (u != s) {
            v = u;
            u = parent[v];
            *path_flow = min(*path_flow, rGraph[u * V + v]);
        }
    }
}

// CUDA kernel for updating residual capacities
__global__ void cuda_update_residual_capacities(int* rGraph, int* parent, int* path_flow, int s, int t) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx == t) {
        int v = idx;
        int u = parent[v];
        while (u != s) {
            v = u;
            u = parent[v];
            rGraph[u * V + v] -= *path_flow;
            rGraph[v * V + u] += *path_flow;
        }
    }
}

// Returns the maximum flow from s to t in the given graph
int fordFulkerson(int* graph, int s, int t) {
    int* rGraph;
    hipMalloc(&rGraph, V * V * sizeof(int));
    hipMemcpy(rGraph, graph, V * V * sizeof(int), hipMemcpyHostToDevice);

    bool* visited;
    hipMalloc(&visited, V * sizeof(bool));
    bool* d_found;
    hipMalloc(&d_found, sizeof(bool));
    int* parent;
    hipMalloc(&parent, V * sizeof(int));

    int max_flow = 0;
    bool found_path;

    do {
        found_path = false;
        hipMemset(visited, 0, V * sizeof(bool));
        hipMemset(d_found, false, sizeof(bool));

        cuda_bfs<<<(V + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(rGraph, visited, parent, t, d_found);
        hipDeviceSynchronize();

        bool h_found;
        hipMemcpy(&h_found, d_found, sizeof(bool), hipMemcpyDeviceToHost);

        if (h_found) {
            int* path_flow;
            hipMalloc(&path_flow, sizeof(int));

            cuda_calculate_path_flow<<<1, 1>>>(rGraph, parent, path_flow, s, t);
            hipDeviceSynchronize();

            int h_path_flow;
            hipMemcpy(&h_path_flow, path_flow, sizeof(int), hipMemcpyDeviceToHost);

            cuda_update_residual_capacities<<<1, 1>>>(rGraph, parent, path_flow, s, t);
            hipDeviceSynchronize();

            max_flow += h_path_flow;
            hipFree(path_flow);
        }
    } while (found_path);

    hipFree(rGraph);
    hipFree(visited);
    hipFree(d_found);
    hipFree(parent);

    return max_flow;
}

int main() {
    std::vector<std::vector<int>> adjMatrix = {
        {667, 367, 300, 367, 300},
        {100, 533, 0, 200, 0},
        {122, 0, 578, 0, 244},
        {50, 100, 0, 283, 183},
        {61, 0, 122, 150, 272}
    };

    int* graph = new int[V * V];
    for (int i = 0; i < V; ++i) {
        for (int j = 0; j < V; ++j) {
            graph[i * V + j] = adjMatrix[i][j];
        }
    }

    int max_flow = fordFulkerson(graph, 0, 4);

    std::cout << "The maximum possible flow is " << max_flow << std::endl;

    delete[] graph;

    return 0;
}
