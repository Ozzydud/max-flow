#include <iostream>
#include <vector>
#include <queue>
#include <hip/hip_runtime.h>

#define V 6 // Assuming V is a constant

__global__ void cudaBFS(int *rGraph, bool *visited, int *parent, int *nextLevel, bool *found, int t) {
    int u = blockIdx.x * blockDim.x + threadIdx.x;
    if (*found || u >= V) return;

    if (visited[u]) {
        int start = u * V;
        for (int v = 0; v < V; ++v) {
            if (!visited[v] && rGraph[start + v] > 0) {
                parent[v] = u;
                if (v == t) {
                    *found = true;
                    return;
                }
                nextLevel[v] = 1;
            }
        }
    }
}

bool bfs(int rGraph[V][V], int s, int t, int parent[V]) {
    int *dev_rGraph, *dev_parent, *dev_nextLevel;
    bool *dev_visited, *dev_found;
    bool found = false;

    int rGraphSize = V * V * sizeof(int);
    int visitedSize = V * sizeof(bool);
    int parentSize = V * sizeof(int);

    hipMalloc((void**)&dev_rGraph, rGraphSize);
    hipMalloc((void**)&dev_visited, visitedSize);
    hipMalloc((void**)&dev_parent, parentSize);
    hipMalloc((void**)&dev_nextLevel, visitedSize);
    hipMalloc((void**)&dev_found, sizeof(bool));

    hipMemcpy(dev_rGraph, rGraph, rGraphSize, hipMemcpyHostToDevice);
    hipMemset(dev_visited, 0, visitedSize);
    hipMemset(dev_nextLevel, 0, visitedSize);
    hipMemset(dev_found, 0, sizeof(bool));

    std::queue<int> q;
    q.push(s);
    hipMemcpy(dev_parent + s, &s, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_visited + s, &s, sizeof(bool), hipMemcpyHostToDevice);

    while (!q.empty() && !found) {
        int u = q.front();
        q.pop();
        cudaBFS<<<(V + 255) / 256, 256>>>(dev_rGraph, dev_visited, dev_parent, dev_nextLevel, dev_found, t);
        hipDeviceSynchronize();

        hipMemcpy(&found, dev_found, sizeof(bool), hipMemcpyDeviceToHost);
        if (found) break;

        hipMemcpy(dev_visited, dev_nextLevel, visitedSize, hipMemcpyDeviceToDevice);
        hipMemset(dev_nextLevel, 0, visitedSize);

        for (int v = 0; v < V; ++v) {
            if (rGraph[u][v] > 0 && !dev_visited[v]) {
                q.push(v);
                hipMemcpy(dev_parent + v, &u, sizeof(int), hipMemcpyHostToDevice);
                hipMemcpy(dev_visited + v, &v, sizeof(bool), hipMemcpyHostToDevice);
            }
        }
    }

    if (found) {
        int current = t;
        while (current != -1) {
            parent[current] = dev_parent[current];
            hipMemcpy(&current, dev_parent + current, sizeof(int), hipMemcpyDeviceToHost);
        }
    }

    hipFree(dev_rGraph);
    hipFree(dev_visited);
    hipFree(dev_parent);
    hipFree(dev_nextLevel);
    hipFree(dev_found);

    return found;
}

int main() {
    int rGraph[V][V] = {
        {0, 16, 13, 0, 0, 0},
        {0, 0, 10, 12, 0, 0},
        {0, 4, 0, 0, 14, 0},
        {0, 0, 9, 0, 0, 20},
        {0, 0, 0, 7, 0, 4},
        {0, 0, 0, 0, 0, 0}
    };

    int parent[V];
    int s = 0, t = 5;
    if (bfs(rGraph, s, t, parent)) {
        std::cout << "Path found from " << s << " to " << t << ":\n";
        for (int i = 0; i < V; ++i) {
            std::cout << i << " <- ";
            if (parent[i] == -1) std::cout << "Source";
            else std::cout << parent[i];
            std::cout << "\n";
        }
    } else {
        std::cout << "No path exists from " << s << " to " << t << "\n";
    }

    return 0;
}
