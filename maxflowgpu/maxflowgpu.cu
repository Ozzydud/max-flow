#include <iostream>
#include <fstream>
#include <vector>

// CUDA libraries
#include <hip/hip_runtime.h>


#define INF 1e9

// BFS
__global__ void cudaBFS (int *row, int *indices, int *data,
                         int source, int sink, int *parent, int *queue, int *flow, int *residual, bool *visited){
    int tid = blockIdx.x * blockDim.x * threadIdx.x; //Finding thread ID
    int vertices = 5;
    if(visited[tid] == false && vertices > tid){ //Mark as visited and add tid to the queue
        queue[tid] = tid;
        visited[tid] = true;
        parent[tid] = -1;
    }

     __syncthreads(); // Not optimal - we need to wait for all threads before we do BFS

     while (!visited[sink] && !visited[source]) { //We keep going as long as we have not visited both sink and source
            // Needs changing to fit with our data ---- ALL OF THE BELOW
            for (int i = row[tid]; i < row[tid + 1]; ++i) {
            int v = indices[i]; // Get the destination vertex
            if (!visited[v] && residual[i] > 0) {
                // Process neighboring vertices
                    queue[v] = tid;
                    visited[v] = true;
                    parent[v] = tid;
            }
        }
         __syncthreads();
     }

}

//AUGMENTED PATHS
__global__ void augmentPath(int *residual, int *parent, int *flow, 
                            int source, int sink){
    int tid = blockIdx.x * blockDim.x * threadIdx.x; //Finding thread ID
    int vertices = 5;
    if(tid<vertices && parent[tid] != -1){ //if == -1, it was not reached in BFS
        int min_flow = INF;
        int current = tid;
        while (current != source) {
            int current_parent = parent[current];
            // Needs changing to follow data structure
            min_flow = min(min_flow, residual[current_parent * vertices + current]);
            current = current_parent;
        }

        current = tid;
        while(current != source){
            int current_parent = parent[current];
            residual[current_parent * vertices + current] -= min_flow;
            residual[current * vertices + current_parent] += min_flow;
            current = current_parent;
        }
        flow[tid] += min_flow;
    }
}


int main() {
    std::cout << "no errors plz";
    return 0;

}
