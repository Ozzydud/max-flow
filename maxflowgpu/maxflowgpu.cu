#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <queue>
#include <hip/hip_runtime.h>

#define V 6 // Assuming V is a constant

__global__ void cudaBFS(int *rGraph, bool *visited, int *parent, int *nextLevel, int *found, int t) {
    int u = blockIdx.x * blockDim.x + threadIdx.x;
    if (*found || u >= V) return;

    if (visited[u]) {
        int start = u * V;
        for (int v = 0; v < V; ++v) {
            if (!visited[v] && rGraph[start + v] > 0) {
                parent[v] = u;
                if (v == t) {
                    *found = 1;
                    return;
                }
                nextLevel[v] = 1;
            }
        }
    }
}

bool bfs(vector<vector<int>>& rGraph, int s, int t, vector<int>& parent) {
    int *dev_rGraph, *dev_parent, *dev_nextLevel, *dev_found;
    bool *dev_visited;
    bool found = false;

    int rGraphSize = V * V * sizeof(int);
    int visitedSize = V * sizeof(bool);
    int parentSize = V * sizeof(int);
    int nextLevelSize = V * sizeof(int);
    int foundSize = sizeof(bool);

    hipMalloc((void**)&dev_rGraph, rGraphSize);
    hipMalloc((void**)&dev_visited, visitedSize);
    hipMalloc((void**)&dev_parent, parentSize);
    hipMalloc((void**)&dev_nextLevel, nextLevelSize);
    hipMalloc((void**)&dev_found, foundSize);

    hipMemcpy(dev_rGraph, rGraph.data(), rGraphSize, hipMemcpyHostToDevice);
    hipMemset(dev_visited, 0, visitedSize);
    hipMemset(dev_nextLevel, 0, nextLevelSize);
    hipMemset(dev_found, 0, foundSize);

    std::queue<int> q;
    q.push(s);
    hipMemcpy(dev_parent + s, &s, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_visited + s, &s, sizeof(bool), hipMemcpyHostToDevice);

    while (!q.empty() && !found) {
        int u = q.front();
        q.pop();
        cudaBFS<<<(V + 255) / 256, 256>>>(dev_rGraph, dev_visited, dev_parent, dev_nextLevel, dev_found, t);
        hipDeviceSynchronize();

        hipMemcpy(&found, dev_found, foundSize, hipMemcpyDeviceToHost);
        if (found) break;

        hipMemcpy(dev_visited, dev_nextLevel, visitedSize, hipMemcpyDeviceToDevice);
        hipMemset(dev_nextLevel, 0, nextLevelSize);

        for (int v = 0; v < V; ++v) {
            if (rGraph[u][v] > 0 && !visited[v]) {
                q.push(v);
                hipMemcpy(dev_parent + v, &u, sizeof(int), hipMemcpyHostToDevice);
                hipMemcpy(dev_visited + v, &v, sizeof(bool), hipMemcpyHostToDevice);
            }
        }
    }

    if (found) {
        int current = t;
        while (current != -1) {
            parent[current] = dev_parent[current];
            hipMemcpy(&current, dev_parent + current, sizeof(int), hipMemcpyDeviceToHost);
        }
    }

    hipFree(dev_rGraph);
    hipFree(dev_visited);
    hipFree(dev_parent);
    hipFree(dev_nextLevel);
    hipFree(dev_found);

    return found;
}

int main() {
    vector<vector<int>> rGraph = {
        {0, 16, 13, 0, 0, 0},
        {0, 0, 10, 12, 0, 0},
        {0, 4, 0, 0, 14, 0},
        {0, 0, 9, 0, 0, 20},
        {0, 0, 0, 7, 0, 4},
        {0, 0, 0, 0, 0, 0}
    };

    vector<int> parent(V, -1);
    int s = 0, t = 5;
    if (bfs(rGraph, s, t, parent)) {
        std::cout << "Path found from " << s << " to " << t << ":\n";
        for (int i = 0; i < V; ++i) {
            std::cout << i << " <- ";
            if (parent[i] == -1) std::cout << "Source";
            else std::cout << parent[i];
            std::cout << "\n";
        }
    } else {
        std::cout << "No path exists from " << s << " to " << t << "\n";
    }

    return 0;
}
