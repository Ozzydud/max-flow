#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <sstream>
#include <string>
#include <vector>
#include <cmath>

#include <bits/stdc++.h>

// CUDA libraries
#include <hip/hip_runtime.h>
#include <>

using namespace std;

#define INF 1e9


void readInput(const char* filename, int total_nodes, int* residual_capacity) {

	ifstream file;
	file.open(filename);

	if (!file) {
        cout <<  "Error reading file!";
        exit(1);
    }

    string line;
    unsigned int source, destination;
    float capacity;

    while (getline(file, line)) {
        if (line.empty()) continue;

        stringstream linestream(line);
        linestream >> source >> destination >> capacity;

        cout << "capacity before " << capacity << " \n";

        int scaledCapacity = static_cast<int>(capacity * 1000);
        residual_capacity[source * total_nodes + destination] = scaledCapacity;

        cout << "capacity after " << residual_capacity[source * total_nodes + destination] << " \n";
    }

    file.close();
}

__global__ void cudaBFS(int *r_capacity, int *parent, int *flow, bool *frontier, bool* visited, int vertices, int sink){
    int Idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (!frontier[sink] && Idx < vertices && frontier[Idx]) {
        frontier[Idx] = false;
        visited[Idx] = true;

        for (int i = 0; i < vertices; i++) {
            if (!frontier[i] && !visited[i] && r_capacity[Idx * vertices + i] > 0) {
                frontier[i] = true;
                parent[i] = Idx;
                flow[i] = min(flow[Idx], r_capacity[Idx * vertices + i]);
            }
        }
    }
}


int main() {
    int total_nodes = 5; // Assuming 5 nodes
    int* residual;

    cout << "test: " << r_capacity << endl;
    // Allocating memory for a square matrix representing the graph
    residual = new int[total_nodes * total_nodes];
    memset(residual, 0, sizeof(int) * total_nodes * total_nodes);

    readInput("cage3.mtx", total_nodes, residual);

    int source = 0;
    int sink = total_nodes - 1; // Assuming sink is the last node

    int* parent = new int[total_nodes];
    int* flow = new int[total_nodes];

    for (int i = 0; i < total_nodes; ++i) {
        parent[i] = -1; // Initialize parent array
        flow[i] = INF;  // Initialize flow array with INF
    }

    // Set initial flow from source to 0
    flow[source] = 0;

    int* d_r_capacity, * d_parent, * d_flow;
    bool* frontier, * visited;

    cout << "test2: " << r_capacity << endl;

    // Allocate memory on device
    hipMalloc((void**)&d_r_capacity, total_nodes * total_nodes * sizeof(int));
    hipMalloc((void**)&d_parent, total_nodes * sizeof(int));
    hipMalloc((void**)&d_flow, total_nodes * sizeof(int));
    hipMalloc((void**)&frontier, total_nodes * sizeof(bool));
    hipMalloc((void**)&visited, total_nodes * sizeof(bool));

    cout << "test3: " << r_capacity << endl;

    // Copy data from host to device
    hipMemcpy(d_r_capacity, residual, total_nodes * total_nodes * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_parent, parent, total_nodes * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_flow, flow, total_nodes * sizeof(int), hipMemcpyHostToDevice);
    hipMemset(frontier, 0, total_nodes * sizeof(bool)); // Initialize to false
    hipMemset(visited, 0, total_nodes * sizeof(bool)); // Initialize to false

    cout << "test4: " << r_capacity << endl;

    bool sink_reachable = true;
    int max_flow = 0;

    while (sink_reachable) {
        sink_reachable = false;

        cout << "test5: " << r_capacity << endl;

        // Initialize frontier array (only the source node is in the frontier)
        hipMemset(frontier + source, 0, sizeof(bool));
        hipMemcpy(frontier + source, &d_flow[source], sizeof(bool), hipMemcpyDeviceToDevice);
        
        // Initialize visited array (all nodes are not visited)
        hipMemset(visited, 0, total_nodes * sizeof(bool));

        // Initialize parent array to -1
        hipMemset(d_parent, -1, total_nodes * sizeof(int));

        int block_size = 256;
        int grid_size = (total_nodes + block_size - 1) / block_size;
        cout << "test6: " << r_capacity << endl;
        // Launch BFS kernel
        cudaBFS<<<grid_size, block_size>>>(d_r_capacity, d_parent, d_flow, frontier, visited, total_nodes, sink);
        hipDeviceSynchronize();
        cout << "test7: " << r_capacity << endl;

        // Check if sink is reachable
        hipMemcpy(&sink_reachable, &frontier[sink], sizeof(bool), hipMemcpyDeviceToHost);

        cout << "test8: " << r_capacity << endl;

        if (sink_reachable) {
            int path_flow = INF;

            cout << "test9: " << r_capacity << endl;

            // Calculate path flow
            for (int v = sink; v != source; v = parent[v]) {
                int u = parent[v];
                path_flow = min(path_flow, residual[u * total_nodes + v]);
            }

            // Update residual capacity and flow along the path
            for (int v = sink; v != source; v = parent[v]) {
                int u = parent[v];
                residual[u * total_nodes + v] -= path_flow;
                residual[v * total_nodes + u] += path_flow;
            }

            max_flow += path_flow;
        }
    }
    cout << "Maximum Flow: " << d_r_capacity << endl;
    cout << "Maximum Flow: " << max_flow << endl;

    // Clean up allocated memory
    delete[] residual;
    delete[] parent;
    delete[] flow;
    hipFree(d_r_capacity);
    hipFree(d_parent);
    hipFree(d_flow);
    hipFree(frontier);
    hipFree(visited);

    return 0;
}