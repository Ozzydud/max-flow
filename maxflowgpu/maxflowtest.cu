#include <iostream>
#include <fstream>
#include <sstream>
#include <string>
#include <vector>
#include <cmath>

#include <bits/stdc++.h>

// CUDA libraries
#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>

using namespace std;

#define INF 1e9

class Edge {
public:
    int source, destination;
    int capacity;

    Edge(int src, int dest, int cap) : source(src), destination(dest), capacity(cap) {}
};

void readInput(const char* filename, int total_nodes, vector<Edge>& edges) {
    ifstream file;
    file.open(filename);

    if (!file) {
        cout <<  "Error reading file!";
        exit(1);
    }

    string line;
    int source, destination;
    int numberOfEdges = 0;

    float capacity;

    while (getline(file, line)) {
        if (line.empty()) continue;

        stringstream linestream(line);

        if (!(linestream >> source >> destination >> capacity)) {
            cerr << "Error parsing line: " << line << endl;
            continue;
        }

        source--;
        destination--;

        int scaledCapacity = static_cast<int>(capacity * 1000);

        Edge edge(source, destination, scaledCapacity);
        edges.push_back(edge);

        numberOfEdges++;
    }
    
    cout << "Number of edges in graph is: " << numberOfEdges << endl;
    file.close();
}

__global__ void cudaBFS(Edge* edges, int num_edges, int* parent, int* flow, bool* frontier, bool* visited, int vertices, int sink, int* locks) {
    int Idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (!frontier[sink] && Idx < vertices && frontier[Idx]) {
        frontier[Idx] = false;
        visited[Idx] = true;

        for (int i = num_edges - 1; i >= 0; i--) { // Traverse edges from bottom to top
        int source = edges[i].source;
        int destination = edges[i].destination;
        int capacity = edges[i].capacity;

        if (source == Idx) {
            if (destination < Idx)
                break;

        if (!frontier[destination] && !visited[destination] && capacity > 0) {
            if (atomicCAS(locks + destination, 0 , 1) == 1 || frontier[destination]) {
                continue;
            }

            frontier[destination] = true;
            locks[destination] = 0;
            parent[destination] = Idx;
            flow[destination] = min(flow[Idx], capacity);
        }
    }
}

    }
}

__global__ void cudaAugment_path(int* parent, bool* do_change_capacity, int total_nodes, Edge* edges, int num_edges, int path_flow) {
    int Idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(Idx < total_nodes && do_change_capacity[Idx]){
        for (int i = 0; i < num_edges; i++) {
            int source = edges[i].source;
            int destination = edges[i].destination;
            if (destination == Idx && source == parent[Idx]) {
                edges[i].capacity -= path_flow;
                break;
            }
        }
    }
}

bool sink_reachable(bool* frontier, int total_nodes, int sink) {
    for (int i = total_nodes-1; i > -1; --i) {
                if(frontier[i]){
                        return i == sink;
                }
        }
        return true;
}

int edmondsKarp(const char* filename, int total_nodes) {
    hipError_t cudaStatus = hipSetDevice(4);
    if (cudaStatus != hipSuccess) {
        std::cerr << "hipSetDevice failed! Do you have a CUDA-capable GPU installed?";
        return 1;
    }

    // Assuming 3534 or 1107 nodes or 11397 or 39082 or 130228

    float avgBFSTime = 0;
    int bfsCounter = 0;
    hipEvent_t startEvent, stopEvent;
    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);


    float avgAUGTime = 0;
    int augCounter = 0;
    hipEvent_t startEvent2, stopEvent2;
    hipEventCreate(&startEvent2);
    hipEventCreate(&stopEvent2);

	

    hipEvent_t start, stop; // Declare start and stop events
    float milliseconds = 0; // Variable to store elapsed time in milliseconds

	    // Initialize CUDA events
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);

    vector<Edge> edges;
    readInput(filename, total_nodes, edges);
    cout << "data read" << endl;

    int source = 0;
    int sink = total_nodes - 1; // Assuming sink is the last node
    int path_flow;
    
    int* parent = new int[total_nodes];
    int* flow = new int[total_nodes];
    bool* frontier = new bool[total_nodes];
    bool* visited = new bool[total_nodes];
    bool* do_change_capacity = new bool[total_nodes];
    
    // Set initial flow from source to 0
    flow[source] = 0;
    int* locks = new int[total_nodes];
    Edge* d_edges;
    int* d_parent, * d_flow, * d_locks;
    bool* d_frontier, * d_visited, * d_do_change_capacity;

    size_t edges_size = edges.size() * sizeof(Edge);
    size_t locks_size = total_nodes * sizeof(int);
    
    // Allocate memory on device
    
    hipMalloc((void**)&d_edges, edges_size);
    hipMalloc((void**)&d_parent, total_nodes * sizeof(int));
    hipMalloc((void**)&d_flow, total_nodes * sizeof(int));
    hipMalloc((void**)&d_frontier, total_nodes * sizeof(bool));
    hipMalloc((void**)&d_visited, total_nodes * sizeof(bool));
    hipMalloc((void**)&d_do_change_capacity, total_nodes * sizeof(bool));
    hipMalloc((void**)&d_locks, locks_size);

    // Copy data from host to device
    
    hipMemcpy(d_edges, edges.data(), edges_size, hipMemcpyHostToDevice);

    bool found_augmenting_path;
    int max_flow = 0;
    int block_size = 512;
    int grid_size = ceil(total_nodes * 1.0 / block_size);

    int counter = 0;
    
    do {
        for (int i = 0; i < total_nodes; ++i) {
            parent[i] = -1; // Initialize parent array
            flow[i] = INF;  // Initialize flow array with INF
            locks[i] = 0;
            if (i == source) {
                frontier[i] = true;
            } else {
                frontier[i] = false;
            }
            visited[i] = false;
            do_change_capacity[i] = false;
        }
       
   
        hipMemcpy(d_parent, parent, total_nodes * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_flow, flow, total_nodes * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_frontier, frontier, total_nodes * sizeof(bool), hipMemcpyHostToDevice);
        hipMemcpy(d_visited, visited, total_nodes * sizeof(bool), hipMemcpyHostToDevice);
        hipMemcpy(d_locks, locks, locks_size, hipMemcpyHostToDevice);
         
        while (!sink_reachable(frontier, total_nodes, sink)) {
            hipEventRecord(startEvent, 0);
            cudaBFS<<<grid_size, block_size>>>(d_edges, edges.size(), d_parent, d_flow, d_frontier, d_visited, total_nodes, sink, d_locks);
            bfsCounter++;
        // Stop recording the event
        hipEventRecord(stopEvent, 0);
        hipEventSynchronize(stopEvent);

        // Calculate elapsed time
        float bfsmili = 0.0f;
        hipEventElapsedTime(&bfsmili, startEvent, stopEvent);
        avgBFSTime += bfsmili;
            hipMemcpy(frontier, d_frontier, total_nodes * sizeof(bool), hipMemcpyDeviceToHost);
        }
        

        found_augmenting_path = frontier[sink];

        if (!found_augmenting_path) {
            break;
        }

        hipMemcpy(flow, d_flow, total_nodes * sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(parent, d_parent, total_nodes * sizeof(int), hipMemcpyDeviceToHost);

        path_flow = flow[sink];
        max_flow += path_flow;

        for (int i = sink; i != source; i = parent[i]) {
            do_change_capacity[i] = true;
        }
       
        hipMemcpy(d_do_change_capacity, do_change_capacity, total_nodes * sizeof(bool), hipMemcpyHostToDevice);

        // Launch augmenting path kernel
        hipEventRecord(startEvent2, 0);
        cudaAugment_path<<<grid_size, block_size>>>(d_parent, d_do_change_capacity, total_nodes, d_edges, edges.size(), path_flow);
            augCounter++;
        // Stop recording the event
        hipEventRecord(stopEvent2, 0);
        hipEventSynchronize(stopEvent2);

        // Calculate elapsed time
        float augmili = 0.0f;
        hipEventElapsedTime(&augmili, startEvent2, stopEvent2);
        avgAUGTime += augmili;
        counter++;
    } while (found_augmenting_path);
    cout << "Max flor is: " << max_flow << endl;
    cout << "Counter is: " << counter << endl;
    cout << "Average BFS time is: " << avgBFSTime / bfsCounter << "ms\n";
    cout << "Total time BFS is: " << avgBFSTime << "ms\n";
    cout << "Average AUG time is " << avgAUGTime << "ms\n";
    cout << "Total AUG time is: " << avgAUGTime / augCounter << "ms\n";

    hipEventRecord(stop);
        hipEventSynchronize(stop);
	    hipEventElapsedTime(&milliseconds, start, stop);
	        cout << "Time for BFS and augmenting path: " << milliseconds << " ms\n";

    // Clean up allocated memory
    delete[] parent;
    delete[] flow;
    delete[] locks;
    delete[] frontier;
    delete[] visited;
    delete[] do_change_capacity;
    hipFree(d_edges);
    hipFree(d_parent);
    hipFree(d_flow);
    hipFree(d_frontier);
    hipFree(d_visited);
    hipFree(d_locks);
    hipFree(d_do_change_capacity);


    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipEventDestroy(stopEvent);
    hipEventDestroy(startEvent);

    hipEventDestroy(stopEvent2);
    hipEventDestroy(startEvent2);

    return 0;
}

int main(){
    -
    edmondsKarp("data/cage11.mtx", 39082);



}
