#include <iostream>
#include <fstream>
#include <sstream>
#include <string>
#include <vector>
#include <cmath>
#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>

using namespace std;

#define INF 1e9

void readInput(const char* filename, int total_nodes, int* residual) {
    ifstream file;
    file.open(filename);

    if (!file) {
        cout << "Error reading file!";
        exit(1);
    }

    string line;
    int source, destination;
    int numberOfEdges = 0;
    float capacity;

    while (getline(file, line)) {
        if (line.empty()) continue;

        stringstream linestream(line);
        if (!(linestream >> source >> destination >> capacity)) {
            cerr << "Error parsing line: " << line << endl;
            continue;
        }

        source--;
        destination--;
        int scaledCapacity = static_cast<int>(capacity * 1000);
        if (!residual) {
            cerr << "Memory allocation failed for residual matrix.";
            exit(EXIT_FAILURE);
        }

        numberOfEdges++;
        residual[source * total_nodes + destination] = scaledCapacity;
    }

    cout << "Number of edges in graph is: " << numberOfEdges << endl;
    file.close();
}

__global__ void cudaBFS_TopDown(int *r_capacity, int *parent, int *flow, bool *frontier, bool* visited, int vertices, int source, int* locks) {
    int Idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (Idx < vertices && frontier[Idx]) {
        frontier[Idx] = false;
        visited[Idx] = true;

        for (int i = 0; i < vertices; i++) {
            if (!frontier[i] && !visited[i] && r_capacity[Idx * vertices + i] > 0) {
                if (atomicCAS(locks + i, 0, 1) == 1 || frontier[i]) {
                    continue;
                }
                frontier[i] = true;
                locks[i] = 0;
                parent[i] = Idx;
                flow[i] = min(flow[Idx], r_capacity[Idx * vertices + i]);
            }
        }
    }
}

__global__ void cudaBFS_BottomUp(int *r_capacity, int *parent, int *flow, bool *frontier, bool* visited, int vertices, int source, int* locks) {
    int Idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (Idx < vertices && !visited[Idx]) {
        for (int i = 0; i < vertices; i++) {
            if (frontier[i] && r_capacity[i * vertices + Idx] > 0) {
                if (atomicCAS(locks + Idx, 0, 1) == 1 || frontier[Idx]) {
                    continue;
                }
                frontier[Idx] = true;
                locks[Idx] = 0;
                parent[Idx] = i;
                flow[Idx] = min(flow[i], r_capacity[i * vertices + Idx]);
                visited[Idx] = true;
            }
        }
    }
}

__global__ void cudaAugment_path(int* parent, bool* do_change_capacity, int total_nodes, int* r_capacity, int path_flow) {
    int Idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (Idx < total_nodes && do_change_capacity[Idx]) {
        r_capacity[Idx * total_nodes + parent[Idx]] -= path_flow;
        r_capacity[parent[Idx] * total_nodes + Idx] += path_flow;
    }
}

bool source_reachable(bool* frontier, int total_nodes, int source) {
    
    for (int i = 0; i <= total_nodes-1; ++i) {
        if (frontier[i]) {
            return i == source;  // Source node is reachable from at least one node in the frontier
        }   
    }
    return true;  // Source node is not reachable from any node in the frontier
}

bool sink_reachable(bool* frontier, int total_nodes, int sink){
    for (int i = total_nodes-1; i > -1; --i) {
                if(frontier[i]){
                        return i == sink;
                }
        }
        return true;
}

float edmondskarp(const char* filename, int total_nodes) {
    hipEvent_t startEvent3, stopEvent3, startEvent3_1, stopEvent3_1;
    hipEventCreate(&startEvent3);
    hipEventCreate(&stopEvent3);
    hipEventCreate(&startEvent3_1);
    hipEventCreate(&stopEvent3_1);
    float partinitmili = 0.0f;
    float initmili = 0.0f;
    float totalInitTime = 0.0f;
    hipEventRecord(startEvent3);

    int* residual;
    float avgBFSTime = 0;
    int bfsCounter = 0;
    hipEvent_t startEvent, stopEvent;
    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);

    hipEvent_t start, stop;
    float milliseconds = 0;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    float avgAUGTime = 0;
    int augCounter = 0;
    hipEvent_t startEvent2, stopEvent2;
    hipEventCreate(&startEvent2);
    hipEventCreate(&stopEvent2);

    try {
        residual = new int[total_nodes * total_nodes]();
    } catch (const std::bad_alloc& e) {
        std::cerr << "Failed to allocate memory for the residual matrix: " << e.what() << std::endl;
        return 1;
    }

    readInput(filename, total_nodes, residual);

    int source = 0;
    int sink = total_nodes - 1;
    int path_flow;

    int* parent = new int[total_nodes];
    int* flow = new int[total_nodes];
    bool* frontier = new bool[total_nodes];
    bool* visited = new bool[total_nodes];
    bool* do_change_capacity = new bool[total_nodes];

    flow[source] = 0;
    flow[sink] = 0;
    int* locks = new int[total_nodes];
    int* d_r_capacity, * d_parent, * d_flow, *d_locks;
    bool* d_frontier, * d_visited, *d_do_change_capacity;

    size_t locks_size = total_nodes * sizeof(int);

    hipMalloc((void**)&d_r_capacity, total_nodes * total_nodes * sizeof(int));
    hipMalloc((void**)&d_parent, total_nodes * sizeof(int));
    hipMalloc((void**)&d_flow, total_nodes * sizeof(int));
    hipMalloc((void**)&d_frontier, total_nodes * sizeof(bool));
    hipMalloc((void**)&d_visited, total_nodes * sizeof(bool));
    hipMalloc((void**)&d_do_change_capacity, total_nodes * sizeof(bool));
    hipMalloc((void**)&d_locks, locks_size);

    hipMemcpy(d_r_capacity, residual, total_nodes * total_nodes * sizeof(int), hipMemcpyHostToDevice);

    bool found_augmenting_path;
    int max_flow = 0;
    int block_size = 512;
    int grid_size = ceil(total_nodes * 1.0 / block_size);

    int counter = 0;
    hipEventRecord(stopEvent3);
    hipEventSynchronize(stopEvent3);
    hipEventElapsedTime(&initmili, startEvent3, stopEvent3);
    totalInitTime += initmili;

    bool use_bottom_up = false;
    do {
        hipEventRecord(startEvent3_1);
        for (int i = 0; i < total_nodes; ++i) {
            parent[i] = -1;
            flow[i] = INF;
            locks[i] = 0;
            frontier[i] = (use_bottom_up && i == sink) || (!use_bottom_up && i == source);
            visited[i] = false;
            do_change_capacity[i] = false;
        }

        hipMemcpy(d_parent, parent, total_nodes * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_flow, flow, total_nodes * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_frontier, frontier, total_nodes * sizeof(bool), hipMemcpyHostToDevice);
        hipMemcpy(d_visited, visited, total_nodes * sizeof(bool), hipMemcpyHostToDevice);
        hipMemcpy(d_locks, locks, locks_size, hipMemcpyHostToDevice);

        hipEventRecord(stopEvent3_1);
        hipEventSynchronize(stopEvent3_1);
        hipEventElapsedTime(&partinitmili, startEvent3_1, stopEvent3_1);
        totalInitTime += partinitmili;

        int old_work = 0;
        int new_work = 0;
        cout << "test5" << endl;
        cout << use_bottom_up << !sink_reachable(frontier, total_nodes, sink) << !use_bottom_up << !source_reachable(frontier, total_nodes, source) << endl;
        while ((use_bottom_up && !sink_reachable(frontier, total_nodes, sink)) || (!use_bottom_up && !source_reachable(frontier, total_nodes, source))) {
            hipEventRecord(startEvent, 0);
            cout << "test0" << endl;
            if (use_bottom_up) {
                cout << "test1" << endl;
                cudaBFS_BottomUp<<<grid_size, block_size>>>(d_r_capacity, d_parent, d_flow, d_frontier, d_visited, total_nodes, source, d_locks);
            } else {
                cout << "test2" << endl;
                cudaBFS_TopDown<<<grid_size, block_size>>>(d_r_capacity, d_parent, d_flow, d_frontier, d_visited, total_nodes, source, d_locks);
            }
            cout << "test3" << endl;
            bfsCounter++;
            hipEventRecord(stopEvent, 0);
            hipEventSynchronize(stopEvent);

            float miliseconds1 = 0;
            hipEventElapsedTime(&miliseconds1, startEvent, stopEvent);
            avgBFSTime += miliseconds1;

            hipMemcpy(frontier, d_frontier, total_nodes * sizeof(bool), hipMemcpyDeviceToHost);
            hipMemcpy(visited, d_visited, total_nodes * sizeof(bool), hipMemcpyDeviceToHost);

            new_work = 0;
            for (int i = 0; i < total_nodes; i++) {
                if (visited[i]) {
                    new_work++;
                }
            }

            if (new_work > 2 * old_work) {
                use_bottom_up = !use_bottom_up;
            }
            old_work = new_work;
        }

        found_augmenting_path = frontier[source];

        if (!found_augmenting_path) {
            break;
        }

        hipMemcpy(flow, d_flow, total_nodes * sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(parent, d_parent, total_nodes * sizeof(int), hipMemcpyDeviceToHost);

        path_flow = flow[source];
        max_flow += path_flow;
        cout << max_flow << endl;

        for (int i = source; i != sink; i = parent[i]) {
            do_change_capacity[i] = true;
        }

        hipMemcpy(d_do_change_capacity, do_change_capacity, total_nodes * sizeof(bool), hipMemcpyHostToDevice);

        hipEventRecord(startEvent2, 0);
        cudaAugment_path<<<grid_size, block_size>>>(d_parent, d_do_change_capacity, total_nodes, d_r_capacity, path_flow);
        augCounter++;
        hipEventRecord(stopEvent2, 0);
        hipEventSynchronize(stopEvent2);

        float augmili = 0.0f;
        hipEventElapsedTime(&augmili, startEvent2, stopEvent2);
        avgAUGTime += augmili;

        counter++;
    } while (counter != 3);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);

    cout << "Time for BFS and augmenting path: " << milliseconds << " ms\n";
    cout << "Average BFS time is: " << avgBFSTime / bfsCounter << "ms\n";
    cout << "Total time BFS is: " << avgBFSTime << "ms\n";
    cout << "Total AUG time is " << avgAUGTime << "ms\n";
    cout << "Average AUG time is: " << avgAUGTime / augCounter << "ms\n";
    cout << "Total init time is: " << totalInitTime << "ms\n";
    cout << "Maximum Flow: " << max_flow << endl;

    delete[] residual;
    delete[] parent;
    delete[] flow;
    delete[] locks;
    delete[] frontier;
    delete[] visited;
    delete[] do_change_capacity;
    hipFree(d_r_capacity);
    hipFree(d_parent);
    hipFree(d_flow);
    hipFree(d_frontier);
    hipFree(d_visited);
    hipFree(d_locks);
    hipFree(d_do_change_capacity);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipEventDestroy(stopEvent);
    hipEventDestroy(startEvent);
    hipEventDestroy(stopEvent2);
    hipEventDestroy(startEvent2);
    hipEventDestroy(stopEvent3);
    hipEventDestroy(startEvent3);
    hipEventDestroy(startEvent3_1);
    hipEventDestroy(stopEvent3_1);

    return milliseconds;
}

int main() {
    float ms = 0;
    cout << "cage3.mtx" << endl; 
    float test = edmondskarp("cage3.mtx", 5);
    for (int i = 0; i < 10; i++) {
        ms += edmondskarp("cage3.mtx", 5);
    }

    float ms2 = 0;
    cout << "cage9.mtx" << endl; 
    test = edmondskarp("data/cage9.mtx", 3534);
    for (int i = 0; i < 10; i++) {
        ms2 += edmondskarp("data/cage9.mtx", 3534);
    }

    float ms3 = 0;
    cout << "cage10.mtx" << endl; 
    test = edmondskarp("data/cage10.mtx", 11397);
    for (int i = 0; i < 10; i++) {
        ms3 += edmondskarp("data/cage10.mtx", 11397);
    }

    float ms4 = 0;
    cout << "cage11.mtx" << endl; 
    test = edmondskarp("data/cage11.mtx", 39082);
    for (int i = 0; i < 10; i++) {
        ms4 += edmondskarp("data/cage11.mtx", 39082);
    }

    cout << "cage3.mtx end with an avg speed of " << ms / 10 << endl; 
    cout << "cage9.mtx end with an avg speed of " << ms2 / 10 << endl; 
    cout << "cage10.mtx end with an avg speed of " << ms3 / 10 << endl; 
    cout << "cage11.mtx end with an avg speed of " << ms4 / 10 << endl; 

    return 0;
}
